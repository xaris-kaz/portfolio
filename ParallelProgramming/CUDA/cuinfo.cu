/*Theocharis_Kazakidis_4679*/
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

/* 
 * Retrieves and prints information for every installed NVIDIA
 * GPU device
 */
void cuinfo_print_devinfo()
{
	int num_devs, i;
	hipDeviceProp_t dev_prop;
	
	hipGetDeviceCount(&num_devs);
	if (num_devs == 0)
	{
		printf("No CUDA devices found.\n");
		return;
	}
	
	for (i = 0; i < num_devs; i++)
	{
        hipGetDeviceProperties(&dev_prop, i);

        printf(" ID Συσκευής: %d\n", i);
        printf(" Όνομα Συσκευής: %s\n", dev_prop.name);
        printf(" Υπολογισμός Ικανότητας CUDA: %d.%d\n", dev_prop.major, dev_prop.minor);
        printf(" Πλήθος SMs: %d\n", dev_prop.multiProcessorCount);
        printf(" Μέγιστο Πλήθος Νημάτων Ανά Μπλοκ: %d\n", dev_prop.maxThreadsPerBlock);
        printf(" Συνολική Global Μνήμη: %f GB\n", dev_prop.totalGlobalMem / (1024.0 * 1024.0 * 1024.0));
        printf(" Συνολική Shared Μνήμη Ανά Μπλοκ: %f MB\n", dev_prop.sharedMemPerBlock / (1024.0 * 1024.0));
        printf(" Εκτιμόμενο Συνολικό Πλήθος Πυρήνων: %d\n",dev_prop.multiProcessorCount * dev_prop.maxThreadsPerMultiProcessor * dev_prop.warpSize);
	}
}

int main()
{
	cuinfo_print_devinfo();
	return 0;
}
